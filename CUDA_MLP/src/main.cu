#include <unistd.h>
#include <iostream>
#include <string>
#include <getopt.h>
#include <cassert>
#include "utils.h"
#include "mnist_reader_less.h"
#include "mlp.h"
#include "functions.h"
#include "cuda_mlp.h"
// #include <hip/hip_runtime.h>

using namespace std;

void train(double learning_rate, int epoch_num, int hidden_dim, const string &dataset_path) {
    printf("Learning rate: %f, epoch number: %d, hidden dimension: %d, dataset path: %s\n", learning_rate, epoch_num, hidden_dim, dataset_path.c_str());
    // Read the MNIST dataset
    auto training_images = mnist::read_mnist_image_file<uint8_t>(dataset_path + "/train-images-idx3-ubyte");
    auto training_labels = mnist::read_mnist_label_file<uint8_t>(dataset_path + "/train-labels-idx1-ubyte");
    auto test_images = mnist::read_mnist_image_file<uint8_t>(dataset_path + "/t10k-images-idx3-ubyte");
    auto test_labels = mnist::read_mnist_label_file<uint8_t>(dataset_path + "/t10k-labels-idx1-ubyte");
    printf("Training images: %zu x %zu\n", training_images.size(), training_images[0].size());
    printf("Training labels: %zu\n", training_labels.size());
    assert(training_images.size() == training_labels.size());
    printf("Test images: %zu x %zu\n", test_images.size(), test_images[0].size());
    printf("Test labels: %zu\n", test_labels.size());
    assert(test_images.size() == test_labels.size());

    // Create a neural network with 784 inputs, 100 hidden neurons and 10 outputs
    MLP mlp(784, hidden_dim, 10);

    // Train the network
    for (int epoch = 0; epoch < epoch_num; epoch++) {
        vector<double> losses;
        for (int i = 0; i < training_images.size(); i++) {
            auto x = training_images[i];
            auto l = training_labels[i];
            vector<double> y(10, 0);
            y[l] = 1;
            auto y_hat = mlp.forward(x);
            auto loss = cross_entropy(y, y_hat);
            losses.push_back(loss);
            if (i % 1000 == 0) {
                double sum = 0;
                for (auto &l: losses) {
                    sum += l;
                }
                double avg_loss = sum / losses.size();
                losses.clear();
                printf("Epoch: %d, Iteration: %d, Loss: %f\n", epoch, i, avg_loss);
            }
            mlp.zero_grad();
            mlp.backward(y, y_hat);
            mlp.update(learning_rate);
        }
    }
}


void train_cuda(double learning_rate, int epoch_num, int hidden_dim, const string &dataset_path) {
    printf("Learning rate: %f, epoch number: %d, hidden dimension: %d, dataset path: %s\n", learning_rate, epoch_num, hidden_dim, dataset_path.c_str());
    // Read the MNIST dataset
    auto training_images = mnist::read_mnist_image_file<uint8_t>(dataset_path + "/train-images-idx3-ubyte");
    auto training_labels = mnist::read_mnist_label_file<uint8_t>(dataset_path + "/train-labels-idx1-ubyte");
    auto test_images = mnist::read_mnist_image_file<uint8_t>(dataset_path + "/t10k-images-idx3-ubyte");
    auto test_labels = mnist::read_mnist_label_file<uint8_t>(dataset_path + "/t10k-labels-idx1-ubyte");
    printf("Training images: %zu x %zu\n", training_images.size(), training_images[0].size());
    printf("Training labels: %zu\n", training_labels.size());
    assert(training_images.size() == training_labels.size());
    printf("Test images: %zu x %zu\n", test_images.size(), test_images[0].size());
    printf("Test labels: %zu\n", test_labels.size());
    assert(test_images.size() == test_labels.size());

    hipError_t err;  

    // Create a neural network with 784 inputs, 100 hidden neurons and 10 outputs
    int input_dim = 784;
    int output_dim = 10;

    MLP_CUDA h_mlp_cuda;
    Init_Host_MLP(&h_mlp_cuda, input_dim, hidden_dim, output_dim);
    printf("Init host success\n");
    double *d_W1, *d_W2, *d_b1, *d_b2, *d_W1_grad, *d_W2_grad, *d_b1_grad, *d_b2_grad, *d_y1, *d_z1, *d_y2, *d_z2;
    Init_Device_MLP(&h_mlp_cuda, &d_W1, &d_W2, &d_b1, &d_b2, &d_W1_grad, &d_W2_grad, &d_b1_grad, &d_b2_grad, &d_y1, &d_z1, &d_y2, &d_z2);       //! double pointer! which means we need to pass double**, not double*
    hipDeviceSynchronize();
    printf("Init device success\n");

    // Train the network
    for (int epoch = 0; epoch < epoch_num; epoch++) {
        vector<double> losses;
        for (int iteration = 0; iteration < training_images.size(); iteration++) {
            // printf("Iteration: %d\n", iteration);
            auto x = training_images[iteration];        // type of x: vector<unsigned char>
            auto l = training_labels[iteration];
            vector<double> y_label(output_dim, 0);
            y_label[l] = 1;
            vector<double> input = vector<double>(x.begin(),x.end());
            
            // Copy input data and labels to device memory
            double *d_input, *d_y_label;
            hipMalloc((void **)&d_input, input.size() * sizeof(double));
            hipMalloc((void **)&d_y_label, y_label.size() * sizeof(double));
            err = hipMemcpy(d_input, input.data(), input.size() * sizeof(double), hipMemcpyHostToDevice);
            if (err != hipSuccess) {  
                printf("Failed to copy input data (x) to device: %s\n", hipGetErrorString(err));  
            }  
            err = hipMemcpy(d_y_label, y_label.data(), y_label.size() * sizeof(double), hipMemcpyHostToDevice);
            if (err != hipSuccess) {  
                printf("Failed to copy input data (y) to device: %s\n", hipGetErrorString(err));  
            }  

            //! forward
            printf("\n-----------------------precheck-----------------------------------\n");
            printf("input dim: %i\n", input_dim);
            printf("hidden dim: %i\n", hidden_dim);
            printf("output dim: %i\n", output_dim);
            // printf("weight W1: \n");
            // for (int i = 0; i < 10; ++i) {printf("%f ", d_W1[i]);}  // only print the first 10 elements
            // //! cannot directly access d_W1 here, because it is in a device, not host
            // printf("\ngrad : \n");
            // for (int i = 0; i < 10; ++i) {printf("%f ", d_W1_grad[i]);}
            // printf("\nz2: \n");
            // for (int i = 0; i < output_dim; ++i) {printf("%f ", d_z2[i]);}

            one_layer_forward_sigmoid_kernel<<<1, hidden_dim>>>(d_input, d_W1, d_b1, d_y1, d_z1, input_dim, hidden_dim);          // input -> first hidden layer
            err = hipGetLastError();  
            if (err != hipSuccess) {  
                printf("Failed to launch forward kernel: %s\n", hipGetErrorString(err));        // 从第二次循环才开始fail
                exit(0); 
            }
            printf("forward 1 success\n");

            one_layer_backward_softmax_kernel<<<1, output_dim>>>(d_y1, d_W2, d_b2, d_y2, d_z2, hidden_dim, output_dim);          // first hidden layer -> output
            softmax_normalization_kernel<<<1, 1>>>(d_z2, output_dim);          // add softmax normalization
            printf("forward 2 success\n");

            // zero grad
            set_zero_matrix_kernel<<<1, hidden_dim * input_dim>>>(d_W1, input_dim, hidden_dim);
            set_zero_matrix_kernel<<<1, hidden_dim>>>(d_b1, 1, hidden_dim);
            set_zero_matrix_kernel<<<1, output_dim * hidden_dim>>>(d_W2, hidden_dim, output_dim);
            set_zero_matrix_kernel<<<1, output_dim>>>(d_b2, 1, output_dim);
            printf("zero grad success\n");

            // backward
            one_layer_backward_softmax_kernel<<<1, output_dim>>>(d_y2, d_z2, d_y_label, d_W2_grad, d_b2_grad, hidden_dim, output_dim);          // output -> first hidden layer
            one_layer_backward_sigmoid_kernel<<<1, hidden_dim>>>(d_y1, d_W1, d_b1_grad, d_W1_grad, d_b1, d_input, input_dim, hidden_dim);          // first hidden layer -> input
            printf("backward success\n");

            // update
            matrix_update_kernel<<<1, hidden_dim * input_dim>>>(d_W1, d_W1_grad, learning_rate, input_dim, hidden_dim);
            matrix_update_kernel<<<1, hidden_dim>>>(d_b1, d_b1_grad, learning_rate, 1, hidden_dim);
            matrix_update_kernel<<<1, output_dim * hidden_dim>>>(d_W2, d_W2_grad, learning_rate, hidden_dim, output_dim);
            matrix_update_kernel<<<1, output_dim>>>(d_b2, d_b2_grad, learning_rate, 1, output_dim);
            printf("update success\n");

            // copy output data from device
            Copy_Device_to_Host(&h_mlp_cuda, d_W1, d_W2, d_b1, d_b2, d_W1_grad, d_W2_grad, d_b1_grad, d_b2_grad, d_y1, d_z1, d_y2, d_z2);

            // compute loss after device result is copied to host
            std::vector<double> y_out = std::vector<double>(h_mlp_cuda.z2, h_mlp_cuda.z2 + output_dim);
            double loss = cross_entropy(y_out, y_label);
            printf("loss: %f\n", loss);
            // Print y_label and h_mlp_cuda.z2
            // printf("y_label: ");        // 正确
            // for (int j = 0; j < y_label.size(); j++) {
            //     printf("%f ", y_label[j]);
            // }
            // printf("\n");

            // printf("h_mlp_cuda.z2: ");      // 全0，不符合预期
            // for (int j = 0; j < h_mlp_cuda.output_dim; j++) {
            //     printf("%f ", h_mlp_cuda.z2[j]);
            // }
            // printf("\n");
            losses.push_back(loss);
            // printf("we got here\n");
            if (iteration % 1000 == 0) {
                double sum = 0;
                for (auto &l: losses) {
                    sum += l;
                }
                double avg_loss = sum / losses.size();
                losses.clear();
                printf("Epoch: %d, Iteration: %d, Loss: %f\n", epoch, iteration, avg_loss);
            }
            hipFree(d_input);
            hipFree(d_y_label);

            // debug: observe one iteration
            exit(0);
        }
    }

    // Free device memory
    Free_Host_MLP(&h_mlp_cuda);
    Free_Device_MLP(d_W1, d_W2, d_b1, d_b2, d_W1_grad, d_W2_grad, d_b1_grad, d_b2_grad, d_y1, d_z1, d_y2, d_z2);
}

static const struct option long_options[] = {
        {"lr",      optional_argument, nullptr, 'l'},
        {"epoch",   optional_argument, nullptr, 'e'},
        {"dataset", optional_argument, nullptr, 'd'},
        {"hidden",  optional_argument, nullptr, 'h'},
        {nullptr,   no_argument,       nullptr, 0}
};

int main(int argc, char *argv[]) {
    double learning_rate = 0.001;
    int epoch_num = 10;
    string dataset_path = "../data";
    int hidden_dim = 100;
    int opt, opt_idx;
    while ((opt = getopt_long(argc, argv, "l:e:d:h:", long_options, &opt_idx)) != -1) {
        switch (opt) {
            case 'l':
                learning_rate = atof(optarg);
                break;
            case 'e':
                epoch_num = atoi(optarg);
                break;
            case 'd':
                dataset_path = optarg;
                break;
            case 'h':
                hidden_dim = atoi(optarg);
                break;
            default:
                break;
        }
    }
    // train(learning_rate, epoch_num, hidden_dim, dataset_path);
    cout << "CUDA version" << endl;
    train_cuda(learning_rate, epoch_num, hidden_dim, dataset_path);
    return 0;
}
