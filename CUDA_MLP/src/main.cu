#include <unistd.h>
#include <iostream>
#include <string>
#include <getopt.h>
#include <cassert>
#include "utils.h"
#include "mnist_reader_less.h"
#include "mlp.h"
#include "cuda_mlp.h"
#include <chrono>

using namespace std;

void train(double learning_rate, int epoch_num, int hidden_dim, const string &dataset_path) {
    printf("Learning rate: %f, epoch number: %d, hidden dimension: %d, dataset path: %s\n", learning_rate, epoch_num, hidden_dim, dataset_path.c_str());
    // Read the MNIST dataset
    auto training_images = mnist::read_mnist_image_file<uint8_t>(dataset_path + "/train-images-idx3-ubyte");
    auto training_labels = mnist::read_mnist_label_file<uint8_t>(dataset_path + "/train-labels-idx1-ubyte");
    auto test_images = mnist::read_mnist_image_file<uint8_t>(dataset_path + "/t10k-images-idx3-ubyte");
    auto test_labels = mnist::read_mnist_label_file<uint8_t>(dataset_path + "/t10k-labels-idx1-ubyte");
    printf("Training images: %zu x %zu\n", training_images.size(), training_images[0].size());
    printf("Training labels: %zu\n", training_labels.size());
    assert(training_images.size() == training_labels.size());
    printf("Test images: %zu x %zu\n", test_images.size(), test_images[0].size());
    printf("Test labels: %zu\n", test_labels.size());
    assert(test_images.size() == test_labels.size());

    // Create a neural network with 784 inputs, 100 hidden neurons and 10 outputs
    MLP mlp(784, hidden_dim, 10);

    // Train the network
    for (int epoch = 0; epoch < epoch_num; epoch++) {
        vector<double> losses;
        for (int i = 0; i < training_images.size(); i++) {
            auto x = training_images[i];
            auto l = training_labels[i];
            vector<double> y(10, 0);
            y[l] = 1;
            auto y_hat = mlp.forward(x);
            auto loss = cross_entropy(y, y_hat);
            losses.push_back(loss);

            if (i % 1000 == 0) {
                double sum = 0;
                for (auto &l: losses) {
                    sum += l;
                }
                double avg_loss = sum / losses.size();
                losses.clear();
                printf("Epoch: %d, Iteration: %d, Loss: %f\n", epoch, i, avg_loss);
            }
            mlp.zero_grad();
            mlp.backward(y, y_hat);
            mlp.update(learning_rate);
        }
    }
}


void train_cuda(double learning_rate, int epoch_num, int hidden_dim, const string &dataset_path) {
    printf("Learning rate: %f, epoch number: %d, hidden dimension: %d, dataset path: %s\n", learning_rate, epoch_num, hidden_dim, dataset_path.c_str());
    // Read the MNIST dataset
    auto training_images = mnist::read_mnist_image_file<uint8_t>(dataset_path + "/train-images-idx3-ubyte");
    auto training_labels = mnist::read_mnist_label_file<uint8_t>(dataset_path + "/train-labels-idx1-ubyte");
    auto test_images = mnist::read_mnist_image_file<uint8_t>(dataset_path + "/t10k-images-idx3-ubyte");
    auto test_labels = mnist::read_mnist_label_file<uint8_t>(dataset_path + "/t10k-labels-idx1-ubyte");
    printf("Training images: %zu x %zu\n", training_images.size(), training_images[0].size());
    printf("Training labels: %zu\n", training_labels.size());
    assert(training_images.size() == training_labels.size());
    printf("Test images: %zu x %zu\n", test_images.size(), test_images[0].size());
    printf("Test labels: %zu\n", test_labels.size());
    assert(test_images.size() == test_labels.size());

    hipError_t err;  

    // Create a neural network with 784 inputs, 100 hidden neurons and 10 outputs
    int input_dim = 784;
    int output_dim = 10;

    MLP_CUDA h_mlp_cuda;
    Init_Host_MLP(&h_mlp_cuda, input_dim, hidden_dim, output_dim);
    printf("Init host success\n");
    double *d_W1, *d_W2, *d_b1, *d_b2, *d_W1_grad, *d_W2_grad, *d_b1_grad, *d_b2_grad, *d_y1, *d_z1, *d_y2, *d_z2;
    Init_Device_MLP(&h_mlp_cuda, &d_W1, &d_W2, &d_b1, &d_b2, &d_W1_grad, &d_W2_grad, &d_b1_grad, &d_b2_grad, &d_y1, &d_z1, &d_y2, &d_z2);       //! double pointer! which means we need to pass double**, not double*
    hipDeviceSynchronize();
    printf("Init device success\n");

    // Train the network
    for (int epoch = 0; epoch < epoch_num; epoch++) {
        vector<double> losses;
        for (int iteration = 0; iteration < training_images.size(); iteration++) {
            // printf("----------------------------------------Iteration: %d------------------------------------------\n", iteration);
            auto x = training_images[iteration];        // type of x: vector<unsigned char>
            auto l = training_labels[iteration];
            vector<double> y_label(output_dim, 0);
            y_label[l] = 1;
            vector<double> input = vector<double>(x.begin(),x.end());
            
            // Copy input data and labels to device memory
            double *d_input, *d_y_label;
            hipMalloc((void **)&d_input, input.size() * sizeof(double));
            hipMalloc((void **)&d_y_label, y_label.size() * sizeof(double));
            err = hipMemcpy(d_input, input.data(), input.size() * sizeof(double), hipMemcpyHostToDevice);
            if (err != hipSuccess) {  
                printf("Failed to copy input data (x) to device: %s\n", hipGetErrorString(err));  
            }  
            err = hipMemcpy(d_y_label, y_label.data(), y_label.size() * sizeof(double), hipMemcpyHostToDevice);
            if (err != hipSuccess) {  
                printf("Failed to copy input data (y) to device: %s\n", hipGetErrorString(err));  
            }  

            //! launch kernel
            int threads_per_block = 256;
            int num_blocks = (hidden_dim + threads_per_block - 1) / threads_per_block;
            one_layer_forward_sigmoid_kernel<<<num_blocks, threads_per_block>>>(d_input, d_W1, d_b1, d_y1, d_z1, input_dim, hidden_dim);          // input -> first hidden layer
            num_blocks = (output_dim + threads_per_block - 1) / threads_per_block;
            // one_layer_forward_softmax_kernel<<<num_blocks, threads_per_block>>>(d_y1, d_W2, d_b2, d_y2, d_z2, hidden_dim, output_dim);          // first hidden layer -> output
            one_layer_forward_softmax_kernel<<<num_blocks, threads_per_block>>>(d_z1, d_W2, d_b2, d_y2, d_z2, hidden_dim, output_dim);          // first hidden layer -> output       //! !!!!d_z1 here, not d_y1!!!
            softmax_normalization_kernel<<<1, 1>>>(d_z2, output_dim);          // add softmax normalization

            // zero grad
            num_blocks = (hidden_dim * input_dim + threads_per_block - 1) / threads_per_block;
            set_zero_matrix_kernel<<<num_blocks, threads_per_block>>>(d_W1_grad, input_dim, hidden_dim);
            num_blocks = (hidden_dim + threads_per_block - 1) / threads_per_block;
            set_zero_matrix_kernel<<<num_blocks, threads_per_block>>>(d_b1_grad, 1, hidden_dim);
            num_blocks = (output_dim * hidden_dim + threads_per_block - 1) / threads_per_block;
            set_zero_matrix_kernel<<<num_blocks, threads_per_block>>>(d_W2_grad, hidden_dim, output_dim);
            num_blocks = (output_dim + threads_per_block - 1) / threads_per_block;
            set_zero_matrix_kernel<<<num_blocks, threads_per_block>>>(d_b2_grad, 1, output_dim);

            // backward
            num_blocks = (output_dim + threads_per_block - 1) / threads_per_block;
            one_layer_backward_softmax_kernel<<<num_blocks, threads_per_block>>>(d_z1, d_z2, d_y_label, d_W2_grad, d_b2_grad, hidden_dim, output_dim);          // output -> first hidden layer
            num_blocks = (hidden_dim + threads_per_block - 1) / threads_per_block;
            one_layer_backward_sigmoid_kernel<<<num_blocks, threads_per_block>>>(d_input, d_y1, d_W2, d_b2_grad, d_W1_grad, d_b1_grad, input_dim, hidden_dim, output_dim);          // first hidden layer -> input

            // update
            num_blocks = (hidden_dim * input_dim + threads_per_block - 1) / threads_per_block;
            matrix_update_kernel<<<num_blocks, threads_per_block>>>(d_W1, d_W1_grad, learning_rate, input_dim, hidden_dim);
            num_blocks = (hidden_dim + threads_per_block - 1) / threads_per_block;
            matrix_update_kernel<<<num_blocks, threads_per_block>>>(d_b1, d_b1_grad, learning_rate, 1, hidden_dim);
            num_blocks = (output_dim * hidden_dim + threads_per_block - 1) / threads_per_block;
            matrix_update_kernel<<<num_blocks, threads_per_block>>>(d_W2, d_W2_grad, learning_rate, hidden_dim, output_dim);
            num_blocks = (output_dim + threads_per_block - 1) / threads_per_block;
            matrix_update_kernel<<<num_blocks, threads_per_block>>>(d_b2, d_b2_grad, learning_rate, 1, output_dim);

            // copy output data from device
            Copy_Device_to_Host(&h_mlp_cuda, d_W1, d_W2, d_b1, d_b2, d_W1_grad, d_W2_grad, d_b1_grad, d_b2_grad, d_y1, d_z1, d_y2, d_z2);

            // compute loss after device result is copied to host
            std::vector<double> y_out = std::vector<double>(h_mlp_cuda.z2, h_mlp_cuda.z2 + output_dim);
            double loss = 0;
            loss = cross_entropy(y_label, y_out);
            losses.push_back(loss);
            if (iteration % 1000 == 0) {
                double sum = 0;
                for (auto &l: losses) {
                    sum += l;
                }
                double avg_loss = sum / losses.size();
                losses.clear();
                printf("Epoch: %d, Iteration: %d, Loss: %f\n", epoch, iteration, avg_loss);
            }
            hipFree(d_input);
            hipFree(d_y_label);
        }
    }

    // Free device memory
    Free_Host_MLP(&h_mlp_cuda);
    Free_Device_MLP(d_W1, d_W2, d_b1, d_b2, d_W1_grad, d_W2_grad, d_b1_grad, d_b2_grad, d_y1, d_z1, d_y2, d_z2);
}

static const struct option long_options[] = {
        {"lr",      optional_argument, nullptr, 'l'},
        {"epoch",   optional_argument, nullptr, 'e'},
        {"dataset", optional_argument, nullptr, 'd'},
        {"hidden",  optional_argument, nullptr, 'h'},
        {nullptr,   no_argument,       nullptr, 0}
};

int main(int argc, char *argv[]) {
    double learning_rate = 0.001;
    int epoch_num = 10;
    string dataset_path = "../data";
    int hidden_dim = 100;
    int opt, opt_idx;
    while ((opt = getopt_long(argc, argv, "l:e:d:h:", long_options, &opt_idx)) != -1) {
        switch (opt) {
            case 'l':
                learning_rate = atof(optarg);
                break;
            case 'e':
                epoch_num = atoi(optarg);
                break;
            case 'd':
                dataset_path = optarg;
                break;
            case 'h':
                hidden_dim = atoi(optarg);
                break;
            default:
                break;
        }
    }

    auto start = std::chrono::steady_clock::now();
    cout << "CPU version" << endl;
    train(learning_rate, epoch_num, hidden_dim, dataset_path);
    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start;
    cout << "Time of CPU version: " << elapsed_seconds.count() << "s\n";

    start = std::chrono::steady_clock::now();
    cout << "CUDA version" << endl;
    train_cuda(learning_rate, epoch_num, hidden_dim, dataset_path);
    end = std::chrono::steady_clock::now();
    elapsed_seconds = end-start;
    cout << "Time of CUDA version: " << elapsed_seconds.count() << "s\n";

    return 0;
}
